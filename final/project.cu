#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void rotateAndReverse(unsigned char* input, unsigned char* output, int width, int height)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    int src_index = i * width + j;
    int dst_index = (height - i - 1) * width + (width - j - 1);
    
    output[dst_index] = 255 - input[src_index];
}

int main(int argc, char** argv)
{
    if (argc != 3) {
        printf("Usage: %s input_image output_image\n", argv[0]);
        return 1;
    }
    
    // Open input image file
    FILE* imagein = fopen(argv[1], "rb");
    if (!imagein) {
        printf("Error: Cannot open input image file.\n");
        return 1;
    }
    
    // Read image header
    int width, height, maxVal;
    fscanf(imagein, "P5\n%d %d\n%d\n", &width, &height, &maxVal);
    if (maxVal != 255) {
        printf("Error: Only 8-bit grayscale images are supported.\n");
        fclose(imagein);
        return 1;
    }
    
    // Allocate memory for the input and output images
    unsigned char* input = (unsigned char*)malloc(width * height);
    unsigned char* output = (unsigned char*)malloc(width * height);
    if (!input || !output) {
        printf("Error: Cannot allocate memory for images.\n");
        fclose(imagein);
        return 1;
    }
    
    // Read input image data
    fread(input, sizeof(unsigned char), width * height, imagein);
    fclose(imagein);
    
    // Allocate device memory for the input and output images
    unsigned char* d_input, * d_output;
    hipMalloc(&d_input, width * height);
    hipMalloc(&d_output, width * height);
    
    // Copy input image from host to device
    hipMemcpy(d_input, input, width * height, hipMemcpyHostToDevice);
    
    // Set up kernel launch parameters
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch kernel to rotate and reverse grayscale image
    rotateAndReverse<<<grid, block>>>(d_input, d_output, width, height);
    
    // Copy output image from device to host
    hipMemcpy(output, d_output, width * height, hipMemcpyDeviceToHost);
    
    // Save output image to disk
    FILE* imageout = fopen(argv[2], "wb");
    fprintf(imageout, "P5\n%d %d\n%d\n", width, height, maxVal);
    fwrite(output, sizeof(unsigned char), width * height, imageout);
    fclose(imageout);
    
    // Free memory
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    
    return 0;
}
